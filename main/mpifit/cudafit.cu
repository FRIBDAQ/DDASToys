#include "hip/hip_runtime.h"
/**
 * @author Ron Fox<fox@nscl.msu.edu>
 * @file cudafit.cu
 * @brief Provide trace fitting using the libucdafit library.
 * @note  We provide call compatible interfaces with lmfit1 and lmfit2, 
 * @note  This fit will not thread due to libcudaoptimize's need for us to 
 *        have global data for the device pointers to the trace.
 */

#include "lmfit.h"             // For the fit extension formats.
#include "reductions.cu"
#include <limits>
#include <ctime>
#include <iostream>
#include <stdexcept>
#include <string>
#include <DE_Optimizer.h>      // Optimizer beast.

// Define the parameter numbers for the fits:

static const unsigned A1 = 0;
static const unsigned K1 = 1;	// rise steepness
static const unsigned K2 = 2;	// exponential decay
static const unsigned X1 = 3;
static const unsigned C  = 4;

static const unsigned P1_NPARAMS = 5;

static const unsigned A2 = 5;
static const unsigned K3 = 6;
static const unsigned K4 = 7;
static const unsigned X2 = 8;

static const unsigned P2_NPARAMS = 9;

/**
 *  Here's why we can't have good things (threadable).  The libcudaoptimizer does not let me
 *  (to my knowledge) pass a parameter to my fitness function so I don't know how to get this
 *  information to it other than making it file scoped which is inherently thread-unsafe.
 */

static unsigned short* d_xCoords;        // trace x coordinates.
static unsigned short* d_yCoords;        // trace y coordinates.
static unsigned        n_tracePoints;  // Number of points in the trace.
static float*          h_pWeights(0);    // Host weights pointer.
static float*          d_pWeights(0);    // Device weights pointer.

/**
 * reportCudaError
 *   Report the most recent Cuda error as an std::runtime_error
 * @param context - describes the error context.
 */
static void
reportCudaError(const char* context)
{
  std::string msg("Error: ");
  msg += context;
  msg += " : ";
  hipError_t status = hipGetLastError();
  msg += hipGetErrorString(status);
  throw std::runtime_error(msg);

}

/**
 * traceToGPU
 *   using the limits and saturation values to suppress some trace points
 *   Generates the x/y coordinates of the tracea that's left.
 * @param trace - raw trace.
 * @param limits - Left/right limits of thet race.
 * @param saturation - saturation values for the trace (values >= to this are eliminated).
 * @return - final number of points:
 */
static unsigned traceToGPU(
   std::vector<uint16_t> trace, std::pair<unsigned, unsigned> limits,
   uint16_t saturation
)
{
  std::vector<uint16_t> xcoords;
  std::vector<uint16_t> ycoords;

  int result(0);
  for (int i = limits.first; i < limits.second; i++) {
    if (trace[i] < saturation) {
      xcoords.push_back(i);
      ycoords.push_back(trace[i]);
      result++;
    }
  }
  // Allocate a pair of unsigned short device arrays:   d_xCoords and d_yCoords
  // and move the data from xcoords and ycoords into them:

  if (hipMalloc(&d_xCoords, xcoords.size()*sizeof(unsigned short)) != hipSuccess) {
    reportCudaError("Allocating GPU memory for trace x-coordinates");
  }
  if (hipMalloc(&d_yCoords, ycoords.size()*sizeof(unsigned short)) != hipSuccess) {
    reportCudaError("Allocating GPU memory for trace y-coordinates");
  }

  if (hipMemcpy(
      d_xCoords, xcoords.data(), xcoords.size()*sizeof(unsigned short), hipMemcpyHostToDevice)
      != hipSuccess) {
    reportCudaError("Moving trace x coordinates into the GPU");
  }
  if (hipMemcpy(
      d_yCoords, ycoords.data(), ycoords.size()*sizeof(unsigned short), hipMemcpyHostToDevice)
      != hipSuccess) {
    reportCudaError("Moving trace y coordinates into the GPU");
  }
  // We'll use weights of 1.0;   This can be modified here:

  h_pWeights = static_cast<float*>(malloc(result * sizeof(float)));
  for (int i =0; i < result; i++) {
    h_pWeights[i] = 1.0;
  }
  if(!hipMalloc(&d_pWeights, result*sizeof(float) != hipSuccess)) {
    reportCudaError("Failed to allocates device weights array");
  }
  if (hipMemcpy(d_pWeights, h_pWeights, result*sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
    reportCudaError("Failed to copy wieghts into the device");
  }

  n_tracePoints = result;
  return result;
}
/**
 *  freeTrace
 *     Release the GPU memory associated with the trace:
 */
static void
freeTrace()
{
  hipFree(d_xCoords);
  hipFree(d_yCoords);
  hipFree(d_pWeights);
  free(h_pWeights);
}
/**
 * logistic - GPU FUNCTION!!!
 *    Evaluate a logistic function for the specified parameters and point.
 *    A logistic function is a function with a sigmoidal shape.  We use it
 *    to fit the rising edge of signals DDAS digitizes from detectors.
 *    See e.g. https://en.wikipedia.org/wiki/Logistic_function for
 *    a discussion of this function.
 *
 * @param A  - Amplitude of the signal.
 * @param k1 - steepness of the signal (related to the rise time).
 * @param x1 - Mid point of the rise of the sigmoid.
 * @param x  = Location at which to evaluate the function.
 * @return double
 */
__device__ float
logistic(float A, float  k, float x1, float x)
{
    return A/(1+expf(-k*(x-x1)));
}

/**
 * decay  - GPU FUNCTION!!!
 *    Signals from detectors usually have a falling shape that approximates
 *    an exponential.  This function evaluates this decay at some point.
 *
 *  @param A1 - amplitude of the signal
 *  @param k1 - Decay time factor f the signal.
 *  @param x1 - Position of the pulse.
 *  @param x  - Where to evaluate the signal.
 *  @return double
 */
__device__ float
decay(float A, float k, float  x1, float x)
{
    return A*(expf(-k*(x-x1)));
}


/**
 * singlePulse -- GPU Function
 *    Evaluate the value of a single pulse in accordance with our
 *    canonical functional form.  The form is a sigmoid rise with an
 *    exponential decay that sits on top of a constant offset.
 *    The exponential decay is turned on with switchOn() above when
 *    x > the rise point of the sigmoid.
 *
 * @param A1  - pulse amplitiude
 * @parm  k1  - sigmoid rise steepness.
 * @param k2  - exponential decay time constant.
 * @param x1  - sigmoid position.
 * @param C   - Constant offset.
 * @param x   - Position at which to evaluat this function
 * @return double
 */
__device__ float
singlePulse(
    float A1, float  k1, float  k2, float x1, float  C, float  x
)
{
    return (logistic(A1, k1, x1, x)  * decay(1.0, k2, x1, x)) // decay term
        + C;                                        // constant.
}
/**
 * doublePulse - GPU FUNCTION!!!
 *    Evaluate the canonical form of a double pulse.  This is done
 *    by summing two single pulses.  The constant term is thrown into the
 *    first pulse.  The second pulse gets a constant term of 0.
 *
 * @param A1   - Amplitude of the first pulse.
 * @param k1   - Steepness of first pulse rise.
 * @param k2   - Decay time of the first pulse.
 * @param x1   - position of the first pulse.
 *
 * @param A2   - Amplitude of the second pulse.
 * @param k3   - Steepness of second pulse rise.
 * @param k4   - Decay time of second pulse.
 * @param x2   - position of second pulse.
 *
 * @param C    - Constant offset the pulses sit on.
 * @param x    - position at which to evaluate the pulse.
 * @return double.
 * 
*/
__device__ float
doublePulse(
    float A1, float k1, float k2, float x1,
    float A2, float k3, float  k4, float  x2,
    float  C, float  x    
)
{
    float  p1 = singlePulse(A1, k1, k2, x1, C, x);
    float  p2 = singlePulse(A2, k3, k4, x2, 0.0, x);
    return p1 + p2;
}

/**
 * chiFitness1  -- GPU FUNCTION!!!!
 *
 *   Computes the chisquare fitness for one point in one solution given that d_fitness
 *   has pulled out what we need.  This fitness is for a single pulse fit.
 *   @param pParams - pointer to this solutions parameters.
 *   @param x       - X coordinate.
 *   @param y       - Y coordinate.
 *   @param wt      - weight for this coordinate (for now unused).
 *   @return float  - square of difference between solution and actual.
 */
__device__
float chiFitness1(const float* pParams, float x, float y, float wt)
{
  // Get the parameters from the fit:

  float a  = pParams[A1];
  float k1 = pParams[K1];
  float k2 = pParams[K2];
  float x1 = pParams[X1];
  float c = pParams[C];

  float fit = singlePulse(a, k1, k2, x1, c, x);
  float d   = (y  - fit);
  return d*d;

  
}
/**
 * chiFitness2 -- GPU FUNCTION
 *   Cmoputes the chi squre fitness contribution for one point in one solution
 *   given that our caller has pulled out what we need:
 *   @param pParams - pointer to this solutions parameters.
 *   @param x       - X coordinate.
 *   @param y       - Y coordinate.
 *   @param wt      - weight for this coordinate (for now unused).
 *   @return float  - square of difference between solution and actual.
 */
__device__
float chiFitness2(const float* pParams, float x, float y, float wt)
{
  // Get the parameters from the fit:

  float a1 = pParams[A1];
  float a2 = pParams[A2];
  float k1 = pParams[K1];
  float k3 = pParams[K3];
  float k2 = pParams[K2];
  float k4 = pParams[K4];
  float x1 = pParams[X1];
  float x2 = pParams[X2];
  float c  = pParams[C];

  float fit = doublePulse(a1, k1, k2, x1, a2, k3, k4, x2, c, x);
  float d   = y - fit;
  return d*d;

}

/**
 * d_fitness1   -- GPU FUNCTION!!!!
 *   This function lives in the GPU and:
 *   - Computes the chi-square contribution for a single point for a single solution 
 *     in the swarm for a single pulse with an offset.
 *   - Uses reduceToSum to sum the chisquare contributions over the entire
 *     trace.
 *   The result is put into the fitness value for our solution.
 *
 *  @param pSolutions - pointer to solutions array in the GPU.
 *  @param pFitnesses - pointer to the array of fitnesse for all solutions in the swarm.
 *  @param nParams    - Number of parameters in the fit (should be 5).
 *  @param nSol       - Number of solutions in the swarm.
 *  @param pXcoords   - Trace xcoordinates array.
 *  @param pYcoords   - Trae y coordinates array.
 *  @param pWeights   - Y weights to apply.
 *  @param nPoints    - Number of points in the trace.
 *
 */
__global__
void d_fitness1(const float* pSolutions, float* pFitnesses, int nParams, int nSol,
	       unsigned short* pXcoords, unsigned short* pYcoords, float* pWeights,
	       int nPoints)
{
  extern __shared__ float sqdiff[];  // Locate the chisqr contribs in shared mem.


  // Figure out which solution and point we're working on.  This is based 
  // on our place in the computation's geometry:

  int swarm = blockIdx.x;
  int solno = blockIdx.y + swarm*nSol; // Our solution.
  int ptno  = threadIdx.x;	      // Our point.

  if ((solno <  nSol*gridDim.x) && (ptno < nPoints)) {
    int ipt = ptno + swarm*nPoints;
    float x = pXcoords[ipt];
    float y = pYcoords[ipt];
    sqdiff[ptno]  = chiFitness1(pSolutions + (solno*nParams), x, y, 1.0);

    // Can't do the fanin sum until all threads have computed:

    __syncthreads();

    reduceToSum<float, MAXPOINTS>(sqdiff, ptno);
    __syncthreads();   // The sum is now done into sqdiff[0]:
    if(ptno == 0) {
      pFitnesses[solno] = sqdiff[0];
    }
  }
  
}
/**
 *  d_fitness2  - GPU FUNCTION!!!
 *    Compute the chisquare fitness for one point of one solution in the swarm.
 *    Once that's done in all threads, we fire off our part of a fan-in parallel
 *    sum over our solution.
 *    Much of what we do is figure out our place in the world so that we can pass
 *    the right stuff to chiFitness2 which does the actual computation.
 *
 *  @param pSolutions - pointer to solutions array in the GPU.
 *  @param pFitnesses - pointer to the array of fitnesse for all solutions in the swarm.
 *  @param nParams    - Number of parameters in the fit (should be 5).
 *  @param nSol       - Number of solutions in the swarm.
 *  @param pXcoords   - Trace xcoordinates array.
 *  @param pYcoords   - Trae y coordinates array.
 *  @param pWeights   - Y weights to apply.
 *  @param nPoints    - Number of points in the trace.
 *
 */
__global__
void d_fitness2(const float* pSolutions, float* pFitnesses, int nParams, int nSol,
	       unsigned short* pXcoords, unsigned short* pYcoords, float* pWeights,
	       int nPoints)
{
  extern __shared__ float sqdiff[];  // Locate the chisqr contribs in shared mem.


  // Figure out which solution and point we're working on.  This is based 
  // on our place in the computation's geometry:

  int swarm = blockIdx.x;
  int solno = blockIdx.y + swarm*nSol; // Our solution.
  int ptno  = threadIdx.x;	      // Our point.

  if ((solno <  nSol*gridDim.x) && (ptno < nPoints)) {
    int ipt = ptno + swarm*nPoints;
    float x = pXcoords[ipt];
    float y = pYcoords[ipt];
    sqdiff[ptno]  = chiFitness2(pSolutions + (solno*nParams), x, y, 1.0);

    // Can't do the fanin sum until all threads have computed:

    __syncthreads();

    reduceToSum<float, MAXPOINTS>(sqdiff, ptno);
    __syncthreads();   // The sum is now done into sqdiff[0]:
    if(ptno == 0) {
      pFitnesses[solno] = sqdiff[0];
    }
  }
  
}


/**
 * h_fitSingle
 *    Invokes the kernel that produces the fitness measure.
 *    The fitness is computed in the GPU and is the chi square.
 *
 * @param solutions - pointer to the Cuda solution set.
 * @param fitnesses - pointer to the current fitness set
 * @param grid      - Computational grid being used.
 * @param block     - Shapes of blocks within the grid.
 */
void
h_fitSingle(
   const CudaOptimize::SolutionSet* solutions, CudaOptimize::FitnessSet* fitnesses,
   dim3 grid, dim3 block
)
{
  const float*   d_solutions = solutions->getDevicePositionsConst();    // Current solutions.
  float*         d_fitnesses = fitnesses->get();                        // Where fitnesses go.

  // Figure out how many warps the fitnesses require:

  int nParams = solutions->getProblemDimension();
  nParams     = (nParams + 31)/32;
  nParams     = nParams*32;

  // Which solution:

  int nsol = solutions->getSolutionNumber();

  // Figure out the bocksize of the computation:

  dim3 myBlockSize(n_tracePoints, 1, 1);
  d_fitness1<<< grid, myBlockSize, n_tracePoints*sizeof(float) >>>(
    d_solutions, d_fitnesses, nParams, nsol, d_xCoords, d_yCoords, d_pWeights, n_tracePoints
  );
  hipDeviceSynchronize();
  if (hipGetLastError() != hipSuccess) {
    reportCudaError("Failed to run single pulsse fitness kernel");
  }

}


/**
 * h_fitDouble
 *   Host part to setup computation of the fitnesses across the swarm for our
 *   fits for a double pulse.  Really this just sets up the
 *   kernel call for fitness2 which does the rest.
 * 
 * @param solutions - pointer to the current Solution set.
 * @param fitnessses - pointer to the current fitness set
 * @param grid      - Computaional grid geometry.
 * @param block     - Shapes of the blocks within the grid.
 */
void
h_fitDouble(
   const CudaOptimize::SolutionSet* solutions, CudaOptimize::FitnessSet* fitnesses,
   dim3 grid, dim3 block
)
{
  const float*   d_solutions = solutions->getDevicePositionsConst();    // Current solutions.
  float*         d_fitnesses = fitnesses->get();                        // Where fitnesses go.

  // Figure out how many warps the fitnesses require:

  int nParams = solutions->getProblemDimension();
  nParams     = (nParams + 31)/32;
  nParams     = nParams*32;

  // Which solution:

  int nsol = solutions->getSolutionNumber();

  // Figure out the bocksize of the computation:

  dim3 myBlockSize(n_tracePoints, 1, 1);
  d_fitness2<<< grid, myBlockSize, n_tracePoints*sizeof(float) >>>(
    d_solutions, d_fitnesses, nParams, nsol, d_xCoords, d_yCoords, d_pWeights, n_tracePoints
  );
  hipDeviceSynchronize();
  if (hipGetLastError() != hipSuccess) {
    reportCudaError("Failed to run single pulsse fitness kernel");
  }
  
}

/**
 * cudafit1
 *   Fit a single pulse to the data:
 * @param pResult - pointer to the resulting parameters.
 * @param trace   - references the raw trace data.
 * @param limits  - Provides the limits over which the trace is done.
 * @param saturation - Defines the FADC saturation level.
 * @param freeTraceWhenDone - if true (default) the trace data is freed from the GPU
 *                     if not it's left allocated.  This allows a double fit to be done
 *                     immediately after with no reallocation/copy.
 */
void
cudafit1(
	 DDAS::fit1Info* pResult, const std::vector<uint16_t>& trace,
	 const std::pair<unsigned, unsigned>& limits,
	 uint16_t saturation = 0xffff, bool freeTraceWhenDone=true
)
{
  size_t nPoints = traceToGPU(trace, limits, saturation);

  // Create and setup the optimizer - fitness function will be done in the device:

  CudaOptimize::DE_Optimizer opt(&h_fitSingle, P1_NPARAMS, 1, 200);   // last parameter the swarmsize?
  opt.setTerminationFlags((CudaOptimize::TERMINATION_FLAGS)(CudaOptimize::TERMINATE_GENS | CudaOptimize::TERMINATE_FIT));
  opt.setGenerations(100); 
  opt.setStoppingFitness(10.0);
  opt.setMutation(CudaOptimize::DE_RANDOM);
  opt.setCrossover(CudaOptimize::DE_BINOMIAL);
  opt.setHostFitnessEvaluation(false);


  // Set constraints on the parameters.

  opt.setBounds(0, A1, make_float2(saturation*10, 0.0));
  opt.setBounds(0, K1, make_float2(500.0, 0.0));
  opt.setBounds(0, K2, make_float2(500.0, 0.0));
  opt.setBounds(0, X1, make_float2(-50.0, nPoints+50));    // Let the positions go a bit before/past the trace.
  opt.setBounds(0, C,  make_float2(saturation/4.0, 0.0));  // 25% full scale offset should be generous.
  
  opt.optimize();

  if (freeTraceWhenDone) freeTrace();

  // Pull out the fit values into the pResult.

  pResult->chiSquare =  opt.getBestFitness(0);
  pResult->fitStatus =  0;
  pResult->iterations = opt.getFunctionEvals();	// closest to an iteration count we have.
  float* pParams      = opt.getBestSolution(0);
  pResult->offset     = pParams[C];
  pResult->pulse.position = pParams[X1];
  pResult->pulse.amplitude= pParams[A1];
  pResult->pulse.steepness= pParams[K1];
  pResult->pulse.decayTime = pParams[K2];


}
/**
 * cudafit2
 *   Two a double pulse fit using libcudaoptimize.
 *
 * @param pResult - pointer to the resulting parameters.
 * @param trace   - references the raw trace data.
 * @param limits  - Provides the limits over which the trace is done.
 * @param saturation - Defines the FADC saturation level.
 * @param traceIsLoaded - if true, the trace is already loaded into the GPU
 *                   from a prior cudafit1 call.  Note that regardless the trace is freed
 *                   after we're run.  The default requires us to copy the trace.
 */
void
cudafit2(
	 DDAS::fit2Info* pResult, const std::vector<uint16_t>& trace,
	 const std::pair<unsigned, unsigned>& limits,
	 uint16_t saturation = 0xffff, bool traceIsLoaded = false
)
{
  // If needed get the trace into the GPU

  size_t nPoints;
  if (traceIsLoaded) {
    nPoints = n_tracePoints;                     // From prior load.
  } else {
    nPoints = traceToGPU(trace, limits, saturation);
  }

  // Set up the optimizer with the fitness done in the GPU:

  CudaOptimize::DE_Optimizer opt(h_fitDouble, P2_NPARAMS, 1, 200);
  opt.setTerminationFlags((CudaOptimize::TERMINATION_FLAGS)(CudaOptimize::TERMINATE_GENS | CudaOptimize::TERMINATE_FIT));
  opt.setGenerations(100); 
  opt.setStoppingFitness(10.0);
  opt.setMutation(CudaOptimize::DE_RANDOM);
  opt.setCrossover(CudaOptimize::DE_BINOMIAL);
  opt.setHostFitnessEvaluation(false);

  // Constrain the parameters - unfortunately we can't constrain x1 < x2 :-(
  // We give corresponding parameters in the second pulse the same constraints.

  opt.setBounds(0, A1, make_float2(saturation*10, 0.0));
  opt.setBounds(0, A2, make_float2(saturation*10, 0.0));
  opt.setBounds(0, K1, make_float2(500.0, 0.0));
  opt.setBounds(0, K3, make_float2(500.0, 0.0));
  opt.setBounds(0, K2, make_float2(500.0, 0.0));
  opt.setBounds(0, K4, make_float2(500.0, 0.0));
  opt.setBounds(0, X1, make_float2(-50.0, nPoints+50));    // Let the positions go a bit before/past the trace.
  opt.setBounds(0, X2, make_float2(-50.0, nPoints+50));    // Let the positions go a bit before/past the trace.
  opt.setBounds(0, C,  make_float2(saturation/4.0, 0.0));  // 25% full scale offset should be generous.

  opt.optimize();

  freeTrace();                                             // Always!!
  
  // We only allowed one case so pull the best fitness and best solution from it:

  pResult->chiSquare = opt.getBestFitness(0);
  pResult->fitStatus = 0;
  pResult->iterations= opt.getFunctionEvals();
  float * pParams    = opt.getBestSolution(0);
  pResult->offset    = pParams[C];

  pResult->pulses[0].position = pParams[X1];
  pResult->pulses[0].amplitude= pParams[A1];
  pResult->pulses[0].steepness= pParams[K1];
  pResult->pulses[0].decayTime = pParams[K2];

  pResult->pulses[1].position = pParams[X2];
  pResult->pulses[1].amplitude= pParams[A2];
  pResult->pulses[1].steepness= pParams[K3];
  pResult->pulses[1].decayTime = pParams[K4];


 
}
