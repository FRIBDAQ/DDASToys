#include "hip/hip_runtime.h"
/*
    This software is Copyright by the Board of Trustees of Michigan
    State University (c) Copyright 2017.

    You may use this software under the terms of the GNU public license
    (GPL).  The terms of this license are described at:

     http://www.gnu.org/licenses/gpl.txt

     Authors:
             Ron Fox
             Giordano Cerriza
	     NSCL
	     Michigan State University
	     East Lansing, MI 48824-1321
*/

/** @file:  CudaFitEngine.cpp
 *  @brief: Provide CUDA fit engines for 1-2 pulse fits.
 *          Note this requires that the cuda compiler be used.
 */


#include "jacobian.h"
#include <stdexcept>
#include <math.h>
// Single pulse fit parameter indices:

static const int P1A_INDEX(0);
static const int P1K1_INDEX(1);
static const int P1K2_INDEX(2);
static const int P1X1_INDEX(3);
static const int P1C_INDEX(4);


// Double pulse fit with all parameters free:

static const int P2A1_INDEX(0);
static const int P2K1_INDEX(1);
static const int P2K2_INDEX(2);
static const int P2X1_INDEX(3);
 
static const int P2A2_INDEX(4);
static const int P2K3_INDEX(5);
static const int P2K4_INDEX(6);
static const int P2X2_INDEX(7);

static const int P2C_INDEX(8);



// From functions.cpp -> device:




/**
 * logistic
 *    Evaluate a logistic function for the specified parameters and point.
 *    A logistic function is a function with a sigmoidal shape.  We use it
 *    to fit the rising edge of signals DDAS digitizes from detectors.
 *    See e.g. https://en.wikipedia.org/wiki/Logistic_function for
 *    a discussion of this function.
 *
 * @param A  - Amplitude of the signal.
 * @param k1 - steepness of the signal (related to the rise time).
 * @param x1 - Mid point of the rise of the sigmoid.
 * @param x  = Location at which to evaluate the function.
 * @return double
 */
__device__
static float
logistic(float A, float k, float x1, float x)
{
    return A/(1+exp(-k*(x-x1)));
}

/**
 * decay
 *    Signals from detectors usually have a falling shape that approximates
 *    an exponential.  This function evaluates this decay at some point.
 *
 *  @param A1 - amplitude of the signal
 *  @param k1 - Decay time factor f the signal.
 *  @param x1 - Position of the pulse.
 *  @param x  - Where to evaluate the signal.
 *  @return double
 */
__device__
static float
decay(float A, float k, float x1, float x)
{
    return A*(exp(-k*(x-x1)));
}

/**
 * singlePulse
 *    Evaluate the value of a single pulse in accordance with our
 *    canonical functional form.  The form is a sigmoid rise with an
 *    exponential decay that sits on top of a constant offset.
 *    The exponential decay is turned on with switchOn() above when
 *    x > the rise point of the sigmoid.
 *
 * @param A1  - pulse amplitiude
 * @parm  k1  - sigmoid rise steepness.
 * @param k2  - exponential decay time constant.
 * @param x1  - sigmoid position.
 * @param C   - Constant offset.
 * @param x   - Position at which to evaluat this function
 * @return double
 */
__device__
static float
singlePulse(
    float A1, float k1, float k2, float x1, float C, float x
)
{
    return (logistic(A1, k1, x1, x)  * decay(1.0, k2, x1, x)) // decay term
        + C;                                        // constant.
}

/**
 * doublePulse
 *    Evaluate the canonical form of a double pulse.  This is done
 *    by summing two single pulses.  The constant term is thrown into the
 *    first pulse.  The second pulse gets a constant term of 0.
 *
 * @param A1   - Amplitude of the first pulse.
 * @param k1   - Steepness of first pulse rise.
 * @param k2   - Decay time of the first pulse.
 * @param x1   - position of the first pulse.
 *
 * @param A2   - Amplitude of the second pulse.
 * @param k3   - Steepness of second pulse rise.
 * @param k4   - Decay time of second pulse.
 * @param x2   - position of second pulse.
 *
 * @param C    - Constant offset the pulses sit on.
 * @param x    - position at which to evaluate the pulse.
 * @return double.
 * 
*/
__device__
static float
doublePulse(
    float A1, float k1, float k2, float x1,
    float A2, float k3, float k4, float x2,
    float C, float x    
)
{
    float p1 = singlePulse(A1, k1, k2, x1, C, x);
    float p2 = singlePulse(A2, k3, k4, x2, 0.0, x);
    return p1 + p2;
}

// Support functions that are in the device:

/**
 * dp1dA
 *    Returns the partial derivative of a single pulse with respect to the
 *    amplitude evaluated at a point
 *
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dA)(x)/w
*/
__device__
static float
dp1dA(float k1, float k2, float x1, float x, float w,
      float erise, float efall)
{
    float d = efall;                      // decay(1.0, k2, x1, x);
    float l = 1.0/(1.0 + erise);              // logistic(1.0, k1, x1, x);
    return d*l / w;
}
/**
 * dp1dk1
 *    Partial of single pulse with respect to the rise time constant k1.
 *
 * @param A - current guess at amplitude.
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dk1)(x)/w
 */
__device__
static float
dp1dk1(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float d1 =   A*efall;               // decay(A, k2, x1, x);  
    float d2 =   erise; //              // decay(1.0, k1, x1,  x);   // part of logistic deriv.
    float num = d1*d2*(x - x1);
    float l   =  1.0/(1.0 + erise);     //  logistic(1.0, k1, x1, x);   
    
    
    return (num*l*l)/w;
}
/**
 * dp1dk2
 *    Partial of a single pulse with respect to the decay time constant.
 * @param A - current guess at amplitude.
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dk2)(x)/w
 */
__device__
static float
dp1dk2(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float d1 = A*efall;                   // decay(A, k2, x1, x);
    float num = d1*(x1 - x);
    float l = 1.0/(1.0 + erise);          // logistic(1.0, k1, x1, x);
    
    return (num*l)/w;
}
/**
 * dp1dx1
 *    Partial of a single pulse with respect to the time at the middle
 *    of the pulse's rise.
 *
 * @param A - current guess at amplitude.
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dk2)(x)/w
 */
__device__
static float
dp1dx1(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float dk1 = erise;                   // = decay(1.0, k1, x1, x);
    float dk2 = efall;                   // decay (1.0, k2, x1, x);
    float l   = 1.0/(1.0 + erise);       // logistic(1.0, k1, x1, x);
    
    float left = A*k2*dk2*l;
    float right = A*k1*dk1*dk2*l*l;
    
    return (left - right)/w;
}
/**
 * dp1dC
 *    Partial derivative of single pulse with respect to the constant term
 *    evaluated at a point.
 *
 * @param A - current guess at amplitude.
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dC)(x)/w
 */
__device__
static float
dp1dC(float A, float k1, float k2, float x1, float x, float w)
{
    
    return 1.0/w;
}


/**
 * The residual and jacobian copmutations are pointwise parallel in the device
 *  (GPU)
 */

/**
 * residual1
 *    Compute the residual for a point in the trace with a single pulse fit.
 *
 * @param tracex  - Pointer to trace x values.
 * @param tracey  - Pointer to trace y values.
 * @param resid   - Pointer to residual values.
 * @param len     - Number of trace elements
 * @param C       - Constant.
 * @param A       - Scale factor.
 * @param k1      - rise-steepeness.
 * @param k2      - decay time.
 * @param x1      - position.
 */
__global__
void residual1(
    void* tx, void* ty, void* res, unsigned len,
    float C, float A, float k1, float k2, float x1)
{
    // Figure out our index... we just don't do anything if it's
    // bigger than len:
    
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
      unsigned short* tracex = static_cast<unsigned short*>(tx);
      unsigned short* tracey = static_cast<unsigned short*>(ty);
      float* resid  = static_cast<float*>(res);
        float x = tracex[i];
        float y = tracey[i];
        
        // Compute the function value.
        
        float value = singlePulse(A, k1, k2, x1, C, x);  // ___device__ function.
        
        // Compute and store the difference:
        
        resid[i] = (value - y);
        
    }  
}
/**
 * jacobian1
 *    Compute the jacobian at a single point of the trace for a single pulse fit.
 *
 * @param tracex - pointer to the trace x coords.
 * @param tracey - pointer to the trace y coords.
 * @param j      - pointer to the jacobian matrix (len*5 elements)
 * @param len    - trace length.
 * @param A       - Scale factor.
 * @param k1      - rise-steepeness.
 * @param k2      - decay time.
 * @param x1      - position.
 */
__global__
void jacobian1(
    void* tx, void* ty, void* jac, unsigned len,
    float A, float k1, float k2, float x1
)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
      unsigned short* tracex = static_cast<unsigned short*>(tx);

      float* j = static_cast<float*>(jac);
        float x = tracex[i];
        
        // Common sub-expression elimination:
        
        float erise = exp(-k1*(x - x1));
        float efall = exp(-k2*(x - x1));
        
        float dA = dp1dA(k1, k2, x1, x, 1.0, erise, efall);
        float dk1= dp1dk1(A, k1, k2, x1, x, 1.0, erise, efall);
        float dk2= dp1dk2(A, k1, k2, x1, x, 1.0, erise, efall);
        float dx = dp1dx1(A, k1, k2, x1, x, 1.0, erise, efall);
        float dC = dp1dC(A, k1, k2, x1, x, 1.0);
        
        // Put these results in the appropriate Jacobian element:
        
        int n = i;
        j[n] = dA;   n += len;
        j[n] = dk1;  n += len;
        j[n] = dk2;  n += len;
        j[n] = dx;   n += len;
        j[n] = dC;
    }
}

/**
 *  The class implementationon is in the host (CPU).
 */

/**
 * constructor
 *   - Allocate the device vectors/matrices.
 *   - push the trace x/y points into the GPU where they stay until we're destroyed.
 */
CudaFitEngine1::CudaFitEngine1(std::vector<std::pair<uint16_t, uint16_t>>& data) :
  FitEngine(data)
{
    // Mashall the trace into x/y arrays.. this lets them be cuda memcpied to the
    // GPU
    
    unsigned m_npts = data.size();
    
    unsigned short x[m_npts];
    unsigned short y[m_npts];
    for (int i =0; i < m_npts; i++) {
        x[i] = data[i].first;
        y[i] = data[i].second;
    }
    
    // The trace:
    
    if (hipMalloc(&m_dXtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Failed to allocated X trace points");
    }
    if (hipMalloc(&m_dYtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Failed to allocatee Y trace points");
    }
    if (hipMemcpy(
            m_dXtrace, x, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
        ) != hipSuccess) {
        throwCudaError("Failed to move trace x coords -> gpu");
    }
    if (hipMemcpy(
            m_dYtrace, y, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
        ) != hipSuccess) {
        throwCudaError("Failed to move trace y coords -> gpu");
    }
    // The residual and jacobians need to be allocated but are filled in by
    // the GPU kernels:
    
    if(hipMalloc(&m_dResiduals, m_npts*sizeof(float)) != hipSuccess) {
        throwCudaError("Failed to allocate residual vector");
    }
    if (hipMalloc(&m_dJacobian, m_npts*5*sizeof(float)) != hipSuccess) {
        throwCudaError("Failed to allocated Jacobian");
    }
}
/**
 * destructor just deallocateds the GPU resources.
 */
CudaFitEngine1::~CudaFitEngine1()
{
    // Not much point in error checking as we're not going to be able to
    // do anything about errors here anyway.
    
    hipFree(m_dXtrace);
    hipFree(m_dYtrace);
    hipFree(m_dResiduals);
    hipFree(m_dJacobian);
}
/**
 * jacobian
 *    Invoke the kernal to do the pointwise parallel jacobian computation.
 *    We use a Y size of 32 and x size of npts+31/32.  That is one warp wide.
 *
 * @param p - Parameter vector.
 * @param j - Jacobian matrix.
 */
void
CudaFitEngine1::jacobian(const gsl_vector* p, gsl_matrix* J)
{
    float A   = gsl_vector_get(p, P1A_INDEX);
    float k1  = gsl_vector_get(p, P1K1_INDEX);
    float k2  = gsl_vector_get(p, P1K2_INDEX);
    float x1  = gsl_vector_get(p, P1X1_INDEX);
    float C   = gsl_vector_get(p, P1C_INDEX);
    
    jacobian1<<<(m_npts+31)/32, 32>>>(
        m_dXtrace, m_dYtrace, m_dJacobian, m_npts,
        A, k1, k2, x1
    );
    // Now we need to pull the jacobian out of the device:
    
    float Jac[m_npts*5];       // we'll do it flat:
    if(
        hipMemcpy(Jac, m_dJacobian, m_npts*5*sizeof(float), hipMemcpyDeviceToHost)
        != hipSuccess
    ) {
        throwCudaError("failed to copy Jacobian from device");
    }
    
    // finally, we have to put the jacobian into the gsl J matrix.
    
    for (int i = 0; i < m_npts; i++) {
        gsl_matrix_set(J, i, 0, Jac[i]);
        gsl_matrix_set(J, i, 1, Jac[i+m_npts]);
        gsl_matrix_set(J, i, 2, Jac[i+(2*m_npts)]);
        gsl_matrix_set(J, i, 3, Jac[i+(3*m_npts)]);
        gsl_matrix_set(J, i, 4, Jac[i+(4*m_npts)]);
    }
}
/**
 * residuals
 *    Triggers a pointwise parallel residual kernel in the
 *    Device and impedance matches that with gsl's requirements.
 *
 *  @param p  - parameter vector.
 *  @param r  - Residual vector.
 */
void
CudaFitEngine1::residuals(const gsl_vector* p, gsl_vector* r)
{
    float A   = gsl_vector_get(p, P1A_INDEX);
    float k1  = gsl_vector_get(p, P1K1_INDEX);
    float k2  = gsl_vector_get(p, P1K2_INDEX);
    float x1  = gsl_vector_get(p, P1X1_INDEX);
    float C   = gsl_vector_get(p, P1C_INDEX);

    residual1<<<(m_npts+31)/32, 32>>>(
        m_dXtrace, m_dYtrace, m_dResiduals, m_npts,
        C, A, k1, k2, x1
    );
    // Fetch out the residuals and push the minto the r vector:
    
    float resids[m_npts];
    if (hipMemcpy(
        resids, m_dResiduals, m_npts*sizeof(float), hipMemcpyDeviceToHost) !=
        hipSuccess) {
        throwCudaError("Failed to pull residuals from GPU");
    }
    // Push the results into r:
    
    for (int i =0; i < m_npts; i++) {
        gsl_vector_set(r, i, resids[i]);
    }
}
/**
 * throwCudaError
 *    - Find the last cuda error
 *    - Make a string out of the message we're passed and the cuda error.
 *    - throw this all as a runtime_error
 *
 *  @param msg - context message.
 */
void
CudaFitEngine1::throwCudaError(const char* msg)
{
    std::string e="Error: ";
    e += msg;
    e += " : ";
    
    hipError_t status = hipGetLastError();
    e += hipGetErrorString(status);
    
    throw std::runtime_error(e);
}

///////////////////////////////////////////////////////////////////////////
// CudaFitEngine2 implementation - double pulse fits.
//

// Device (GPU) kernels needed:

/**
 * residual2
 *   Computes the two pulse residual pointwise parallel:
 *
 *   @param xc   - Xcoordinates of trace.
 *   @param yc   - Ycoordinates of trace.
 *   @param r    - Residuals to compute.
 *   @param npts - Number of trace points.
 *   @param C    - Constant offset fit parameter.
 *   @param A1   - Scale factor for pulse1.
 *   @param k11  - K1 for pulse 1.
 *   @param k12  - K2 for pulse 1.
 *   @param x1   - position of pulse 1
 *   @param A2   - Scale factof for pulse 2.
 *   @param k21  - k1 for pulse 2.
 *   @param k22  - k2 for pulse 2.
 *   @param x2   - position of pulse 2.
 */
__global__
void residual2(
    void* xtc, void* ytc, void* res, unsigned npts,
    float C,
    float A1, float k11, float k12, float x1,
    float A2, float k21, float k22, float x2
)
{
    // compute our index and only do anything if its < npts:
    
    int i  = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npts) {
      unsigned short* xc = static_cast<unsigned short*>(xtc);
      unsigned short* yc = static_cast<unsigned short*>(ytc);
      float* r = static_cast<float*>(res);
      float x = xc[i];
        float y = yc[i];
        float fit = doublePulse(A1, k11, k12, x1, A2, k21, k22, x2, C, x);
        r[i] = fit - y;
    }
}

/**
 * jacobian2
 *    Compute the 2 pulse jacobian on a point of the pulse
 *    The jacobian matrix is an npts x 9 matrix.
 *
 * @param xc - x coordinates of the trace.
 * @param j  - jacobian matrix.
 * @param npts - Number of points in the fit.
 * @param A1, k1, k2, x1  - FIt parameters for first pulse.
 * @param A2, k3, k4, x2  - Fit parameters for the second pulse.
 * @param C               - constant term of the fit.
 * 
 */
__global__
void jacobian2(
    void* xtc,  void* jac, unsigned npts,
    float A1, float k1, float k2, float x1,
    float A2, float k3, float k4, float x2,
    float C
)
{
    // figure out which point we're doing and compute if it's in the range
    // of the trace:
    
    int i  = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npts) {
      unsigned short* xc = static_cast<unsigned short*>(xtc);
      float* j = static_cast<float*>(jac);
      
        // now the jacobian elements:
        
        int k = i;            // We'll increment this by npts for each j element
        float x = xc[i];
        
        // Common subexpression elmiination between functions:
        
        float erise1 = exp(-k1*(x - x1));
        float efall1 = exp(-k2*(x - x1));
        
        float erise2 = exp(-k3*(x - x2));
        float efall2 = exp(-k4*(x - x2));
        
        // Pulse 1 elements.
        
        j[k] = dp1dA(k1, k2, x1, x, 1.0, erise1, efall1);      k += npts;
        j[k] = dp1dk1(A1, k1, k2, x1, x, 1.0, erise1, efall1); k += npts;
        j[k] = dp1dk2(A1, k1, k2, x1, x, 1.0, erise1, efall1); k += npts;
        j[k] = dp1dx1(A1, k1, k2, x1, x, 1.0, erise1, efall1); k += npts;
        
        // Pulse 2 elements.
        
        j[k] = dp1dA(k3, k4,x2,x, 1.0, erise2, efall2);        k += npts;
        j[k] = dp1dk1(A2, k3, k4, x2, x, 1.0, erise2, efall2); k += npts;
        j[k] = dp1dk2(A2, k3, k4, x2, x, 1.0, erise2, efall2); k += npts;
        j[k] = dp1dx1(A2, k3, k4, x2, x, 1.0, erise2, efall2); k += npts;
        
        // constant element.
        
        j[k] = 1.0;
    }
}
////////////////////
// Host class implementation:

/**
 * constructor
 *   - Allocate the GPU resources:
 *     *   trace x array
 *     *   trace y array.
 *     *   residual array.
 *     *   Jacobian vector (m_npts * 9)
 *   - Move the trace into the GPU where it stays for all iterations of the fit.
 *
 * @param data - the trace data in x/y pairs.
 */
CudaFitEngine2::CudaFitEngine2(std::vector<std::pair<uint16_t, uint16_t>>&  data) :
  FitEngine(data)
{
    // Make separate x/y arrays from the data:
    
    m_npts = data.size();
    unsigned short x[m_npts];
    unsigned short y[m_npts];
    
    // Allocate the trace arrays and move the trace in:
    
    if (hipMalloc(&m_dXtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Unable to allocate GPU x trace array");
    }
    if (hipMalloc(&m_dYtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Unable to allocate CPU y trace array");
    }
    
    if(
        hipMemcpy(
            m_dXtrace, x, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
        ) != hipSuccess
    ) {
        throwCudaError("Unable to move x coords of trace -> GPU");
    }
    if(hipMemcpy(
        m_dYtrace, y, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
    ) != hipSuccess ) {
        throwCudaError("Unable to move y coords of trace -> GPU");
    }
     // Allocate the residuals and jacobian:
     
    if(hipMalloc(&m_dResiduals, m_npts*sizeof(float)) != hipSuccess) {
        throwCudaError("Unable to allocate residual array in GPU");
    }
    if (hipMalloc(&m_dJacobian, m_npts*9*sizeof(float)) != hipSuccess) {
        throwCudaError("Unable to allocated jacobian matrix in GPU");
    }
}
/**
 * destructor just frees the device blocks
 */
CudaFitEngine2::~CudaFitEngine2()
{
    // No point in looking for errors since we don't know how to recover:
    
    hipFree(m_dXtrace);
    hipFree(m_dYtrace);
    hipFree(m_dResiduals);
    hipFree(m_dJacobian);
}
/**
 * jacobian
 *    Marshall the parameter and call the jacobian2 kernel.  Then
 *    pull the jacobian matrix out of the GPU and marshall it back into
 *    the gsl Jacobian matrix.
 *
 * @param p   - parameter vector from gsl.
 * @param j   - jacobian matrix to output.
 * @note we organize the computing into 32 thread blocks because there are 32 thread
 *       per warp.
 */
void
CudaFitEngine2::jacobian(const gsl_vector* p, gsl_matrix* j)
{
    // Fish the current fit parameters from p:
    
    float A1    = gsl_vector_get(p, P2A1_INDEX);   // Pulse 1.
    float k1    = gsl_vector_get(p, P2K1_INDEX);
    float k2    = gsl_vector_get(p, P2K2_INDEX);
    float x1    = gsl_vector_get(p, P2X1_INDEX);
    
    
    float A2    = gsl_vector_get(p, P2A2_INDEX);   // Pulse 2.
    float k3    = gsl_vector_get(p, P2K3_INDEX);
    float k4    = gsl_vector_get(p, P2K4_INDEX);
    float x2    = gsl_vector_get(p, P2X2_INDEX);
    
    float C     = gsl_vector_get(p, P2C_INDEX);    // constant.
    
    jacobian2<<<(m_npts + 31)/32, 32>>>(
        m_dXtrace, m_dJacobian, m_npts,
        A1, k1, k2, x1,
        A2, k3, k4, x2,
        C
    );
    
    // Fetch the jacobian and marshall it into j.
    
    float jac[m_npts*9];
    if (hipMemcpy(jac, m_dJacobian, m_npts*9*sizeof(float), hipMemcpyDeviceToHost)
        != hipSuccess) {
        throwCudaError("Failed to fetch 2 pulse jacobian from gpu");
    }
    
    for (int i =0; i < m_npts; i++) {
        int k = i;
        gsl_matrix_set(j, i, 0, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 1, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 2, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 3, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 4, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 5, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 6, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 7, jac[k]); k += m_npts;
        gsl_matrix_set(j, i, 8, jac[k]); k += m_npts;    
    }
}
/**
 * residuals
 *    Fire off the kernel to compute the pointwise residuals.
 *
 * @param p - fit parameters
 * @param r - residuals
 */
void
CudaFitEngine2::residuals(const gsl_vector* p, gsl_vector* r)
{
    // Pull out the current fit parameters:
    
    float A1    = gsl_vector_get(p, P2A1_INDEX);   // Pulse 1.
    float k1    = gsl_vector_get(p, P2K1_INDEX);
    float k2    = gsl_vector_get(p, P2K2_INDEX);
    float x1    = gsl_vector_get(p, P2X1_INDEX);
    
    
    float A2    = gsl_vector_get(p, P2A2_INDEX);   // Pulse 2.
    float k3    = gsl_vector_get(p, P2K3_INDEX);
    float k4    = gsl_vector_get(p, P2K4_INDEX);
    float x2    = gsl_vector_get(p, P2X2_INDEX);
    
    float C     = gsl_vector_get(p, P2C_INDEX);    // constant.
 
    // Fire off the kernel to do all this in pointwise parallel.
    
    residual2<<<(m_npts+31)/32,  32>>>(
        m_dXtrace, m_dYtrace, m_dResiduals, m_npts,
        A1, k1, k2, x1, A2, k3, k4, x2, C
    );
    
    // Now we pull out the residuals vector and put it into r:
    
    float residuals[m_npts];
    if (hipMemcpy(
        residuals, m_dResiduals, m_npts*sizeof(float), hipMemcpyDeviceToHost
        ) != hipSuccess) {
        throwCudaError("Unable to fetch residuals from GPU");
    }
    
    for (int i =0; i < m_npts; i++) {
        gsl_vector_set(r, i, residuals[i]);
    }
}
/**
 * throwCudaError
 *     See this method in CudaFitEngine1 - here's a source for factorization
 *     into a base class...along with the allocation of the trace and residual
 *     as well as the push of the trace into the GPU.
 * @param msg - message used to construct the exception messgae.
 */
void
CudaFitEngine2::throwCudaError(const char* msg)
{
    std::string e="Error: ";
    e += msg;
    e += " : ";
    
    hipError_t status = hipGetLastError();
    e += hipGetErrorString(status);
    
    throw std::runtime_error(e);    
}

