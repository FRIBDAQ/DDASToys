#include "hip/hip_runtime.h"
/*
    This software is Copyright by the Board of Trustees of Michigan
    State University (c) Copyright 2017.

    You may use this software under the terms of the GNU public license
    (GPL).  The terms of this license are described at:

     http://www.gnu.org/licenses/gpl.txt

     Authors:
             Ron Fox
             Giordano Cerriza
	     NSCL
	     Michigan State University
	     East Lansing, MI 48824-1321
*/

/** @file:  CudaFitEngine.cpp
 *  @brief: Provide CUDA fit engines for 1-2 pulse fits.
 *          Note this requires that the cuda compiler be used.
 */


#include "jacobian.h"
#include <stdexcept>

// From functions.cpp -> device:




/**
 * logistic
 *    Evaluate a logistic function for the specified parameters and point.
 *    A logistic function is a function with a sigmoidal shape.  We use it
 *    to fit the rising edge of signals DDAS digitizes from detectors.
 *    See e.g. https://en.wikipedia.org/wiki/Logistic_function for
 *    a discussion of this function.
 *
 * @param A  - Amplitude of the signal.
 * @param k1 - steepness of the signal (related to the rise time).
 * @param x1 - Mid point of the rise of the sigmoid.
 * @param x  = Location at which to evaluate the function.
 * @return double
 */
__device__
static float
logistic(float A, float k, float x1, float x)
{
    return A/(1+exp(-k*(x-x1)));
}

/**
 * decay
 *    Signals from detectors usually have a falling shape that approximates
 *    an exponential.  This function evaluates this decay at some point.
 *
 *  @param A1 - amplitude of the signal
 *  @param k1 - Decay time factor f the signal.
 *  @param x1 - Position of the pulse.
 *  @param x  - Where to evaluate the signal.
 *  @return double
 */
__device__
static float
decay(float A, float k, float x1, float x)
{
    return A*(exp(-k*(x-x1)));
}

/**
 * singlePulse
 *    Evaluate the value of a single pulse in accordance with our
 *    canonical functional form.  The form is a sigmoid rise with an
 *    exponential decay that sits on top of a constant offset.
 *    The exponential decay is turned on with switchOn() above when
 *    x > the rise point of the sigmoid.
 *
 * @param A1  - pulse amplitiude
 * @parm  k1  - sigmoid rise steepness.
 * @param k2  - exponential decay time constant.
 * @param x1  - sigmoid position.
 * @param C   - Constant offset.
 * @param x   - Position at which to evaluat this function
 * @return double
 */
__device__
static float
singlePulse(
    float A1, float k1, float k2, float x1, float C, float x
)
{
    return (logistic(A1, k1, x1, x)  * decay(1.0, k2, x1, x)) // decay term
        + C;                                        // constant.
}

/**
 * doublePulse
 *    Evaluate the canonical form of a double pulse.  This is done
 *    by summing two single pulses.  The constant term is thrown into the
 *    first pulse.  The second pulse gets a constant term of 0.
 *
 * @param A1   - Amplitude of the first pulse.
 * @param k1   - Steepness of first pulse rise.
 * @param k2   - Decay time of the first pulse.
 * @param x1   - position of the first pulse.
 *
 * @param A2   - Amplitude of the second pulse.
 * @param k3   - Steepness of second pulse rise.
 * @param k4   - Decay time of second pulse.
 * @param x2   - position of second pulse.
 *
 * @param C    - Constant offset the pulses sit on.
 * @param x    - position at which to evaluate the pulse.
 * @return double.
 * 
*/
__device__
static float
doublePulse(
    float A1, float k1, float k2, float x1,
    float A2, float k3, float k4, float x2,
    float C, float x    
)
{
    float p1 = singlePulse(A1, k1, k2, x1, C, x);
    float p2 = singlePulse(A2, k3, k4, x2, 0.0, x);
    return p1 + p2;
}

// Support functions that are in the device:

/**
 * dp1dA
 *    Returns the partial derivative of a single pulse with respect to the
 *    amplitude evaluated at a point
 *
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dA)(x)/w
*/
__device__
static float
dp1dA(float k1, float k2, float x1, float x, float w,
      float erise, float efall)
{
    float d = efall;                      // decay(1.0, k2, x1, x);
    float l = 1.0/(1.0 + erise);              // logistic(1.0, k1, x1, x);
    return d*l / w;
}
/**
 * dp1dk1
 *    Partial of single pulse with respect to the rise time constant k1.
 *
 * @param A - current guess at amplitude.
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dk1)(x)/w
 */
__device__
static float
dp1dk1(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float d1 =   A*efall;               // decay(A, k2, x1, x);  
    float d2 =   erise; //              // decay(1.0, k1, x1,  x);   // part of logistic deriv.
    float num = d1*d2*(x - x1);
    float l   =  1.0/(1.0 + erise);     //  logistic(1.0, k1, x1, x);   
    
    
    return (num*l*l)/w;
}
/**
 * dp1dk2
 *    Partial of a single pulse with respect to the decay time constant.
 * @param A - current guess at amplitude.
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dk2)(x)/w
 */
__device__
static float
dp1dk2(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float d1 = A*efall;                   // decay(A, k2, x1, x);
    float num = d1*(x1 - x);
    float l = 1.0/(1.0 + erise);          // logistic(1.0, k1, x1, x);
    
    return (num*l)/w;
}
/**
 * dp1dx1
 *    Partial of a single pulse with respect to the time at the middle
 *    of the pulse's rise.
 *
 * @param A - current guess at amplitude.
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dk2)(x)/w
 */
__device__
static float
dp1dx1(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float dk1 = erise;                   // = decay(1.0, k1, x1, x);
    float dk2 = efall;                   // decay (1.0, k2, x1, x);
    float l   = 1.0/(1.0 + erise);       // logistic(1.0, k1, x1, x);
    
    float left = A*k2*dk2*l;
    float right = A*k1*dk1*dk2*l*l;
    
    return (left - right)/w;
}
/**
 * dp1dC
 *    Partial derivative of single pulse with respect to the constant term
 *    evaluated at a point.
 *
 * @param A - current guess at amplitude.
 * @param k1 - current guess at rise steepness param (log(81)/risetime90).
 * @param k2 - current guess at the decay time constant.
 * @param x1 - Current guess at pulse position.
 * @param x  - X at which to evaluate all this.
 * @param w  - weight for the point 
 * @return double - Value of (dP1/dC)(x)/w
 */
__device__
static float
dp1dC(float A, float k1, float k2, float x1, float x, float w)
{
    
    return 1.0/w;
}


/**
 * The residual and jacobian copmutations are pointwise parallel in the device
 *  (GPU)
 */

/**
 * residual1
 *    Compute the residual for a point in the trace with a single pulse fit.
 *
 * @param tracex  - Pointer to trace x values.
 * @param tracey  - Pointer to trace y values.
 * @param resid   - Pointer to residual values.
 * @param len     - Number of trace elements
 * @param C       - Constant.
 * @param A       - Scale factor.
 * @param k1      - rise-steepeness.
 * @param k2      - decay time.
 * @param x1      - position.
 */
__global__
void residual1(
    unsigned short* tracex, unsigned short* tracey, float* resid, unsigned len,
    float C, float A, float k1, float k2, float x1)
{
    // Figure out our index... we just don't do anything if it's
    // bigger than len:
    
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
        float x = tracex[i];
        float y = tracey[i];
        
        // Compute the function value.
        
        float value = singlePulse(A, k1, k2, x1, C, x);  // ___device__ function.
        
        // Compute and store the difference:
        
        resid[i] = (value - y);
        
    }  
}
/**
 * jacobian1
 *    Compute the jacobian at a single point of the trace for a single pulse fit.
 *
 * @param tracex - pointer to the trace x coords.
 * @param tracey - pointer to the trace y coords.
 * @param j      - pointer to the jacobian matrix (len*5 elements)
 * @param len    - trace length.
 * @param A       - Scale factor.
 * @param k1      - rise-steepeness.
 * @param k2      - decay time.
 * @param x1      - position.
 */
__global__
void jacobian1(
    unsigned short* tracex, unsigned short* tracey, float* j, unsigned len,
    float A, float k1, float k2, x1
)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
        float x = tracex[i];
        
        // Common sub-expression elimination:
        
        float erise = exp(-k1*(xi - x1));
        float efall = exp(-k2*(xi - x1));
        
        float dA = dp1dA(k1, k2, x1, x, 1.0, erise, efall);
        float dk1= dp1k1(A, k1, k2, x1, x, 1.0, erise, efall);
        float dk2= dp1k2(A, k1, k2, x1m xm 1,9m erise, efall);
        float dx = dp1dx1(A, k1, k2, x1, x, 1.0, erise, efall);
        float dC = dp1dC(a, k1, k2, x1, x, 1.0);
        
        // Put these results in the appropriate Jacobian element:
        
        int n = i;
        j[n] = dA;   n += len;
        j[n] = dk1;  n += len;
        j[n] = dk2;  n += len;
        j[n] = dx;   n += len;
        j[n] = dC;
    }
}

/**
 *  The class implementationon is in the host (CPU).
 */

/**
 * constructor
 *   - Allocate the device vectors/matrices.
 *   - push the trace x/y points into the GPU where they stay until we're destroyed.
 */
CudaFitEngine1::CudaFitEngine1(std::vector<std::pair<uint16_t, uint16_t>>& data)
{
    // Mashall the trace into x/y arrays.. this lets them be cuda memcpied to the
    // GPU
    
    unsigned m_npts = data.size();
    
    unsigned short x[m_npts];
    unsigned short y[m_npts];
    for (int i =0; i < m_npts; i++) {
        x[i] = data[i].first;
        y[i] = data[i].second;
    }
    
    // The trace:
    
    if (hipMalloc(&m_dxTrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Failed to allocated X trace points");
    }
    if (hipMalloc(&m_dYtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Failed to allocatee Y trace points");
    }
    if (hipMemcpy(
            m_dXtrace, x, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
        ) != hipSuccess) {
        throwCudaError("Failed to move trace x coords -> gpu");
    }
    if (hipMemcpy(
            m_dYtrace, y, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
        ) != hipSuccess) {
        throwCudaError("Failed to move trace y coords -> gpu");
    }
    // The residual and jacobians need to be allocated but are filled in by
    // the GPU kernels:
    
    if(hipMalloc(&m_dResiduals, m_npts*sizeof(float)) != hipSuccess) {
        throwCudaError("Failed to allocate residual vector");
    }
    if (hipMalloc(&m_dJacobian, m_npts*5*sizeof(float)) != hipSuccess) {
        throwCudaError("Failed to allocated Jacobian");
    }
}
/**
 * destructor just deallocateds the GPU resources.
 */
CudaFitEngine1::~CudaFitEngine1()
{
    // Not much point in error checking as we're not going to be able to
    // do anything about errors here anyway.
    
    CudaFree(m_dXtrace);
    CudaFree(m_dYtrace);
    CudaFree(m_dResiduals);
    CudaFree(m_dJacobian);
}
/**
 * jacobian
 *    Invoke the kernal to do the pointwise parallel jacobian computation.
 *    We use a Y size of 32 and x size of npts+31/32.  That is one warp wide.
 *
 * @param p - Parameter vector.
 * @param j - Jacobian matrix.
 */
void
CudaFitEngine1::jacobian(const gsl_vector* p, gsl_matrix* J)
{
    float A   = gsl_vector_get(p, P1A_INDEX);
    float k1  = gsl_vector_get(p, P1K1_INDEX);
    float k2  = gsl_vector_get(p, P1K2_INDEX);
    float x1  = gsl_vector_get(p, P1X1_INDEX);
    float C   = gsl_vector_get(p, P1C_INDEX);
    
    jacobian1<<<(m_npts+31)/32 32>>>(
        m_dXtrace, m_dYtrace, m_dJacobian, m_npts,
        A, k1, k2, x1
    );
    // Now we need to pull the jacobian out of the device:
    
    float Jac[npts*5];       // we'll do it flat:
    if(
        hipMemcpy(Jac, m_dJacobian, npts*5*sizeof(float), hipMemcpyDeviceToHost)
        != hipSuccess
    ) {
        throwCudaError("failed to copy Jacobian from device");
    }
    
    // finally, we have to put the jacobian into the gsl J matrix.
    
    for (int i = 0; i < m_npts; i++) {
        gsl_matrix_set(J, i, 0, Jac[i]);
        gsl_matrix_set(J, i, 1, Jac[i+m_npts]);
        gsl_matrix_set(J, i, 2, Jac[i+(2*m_npts)]);
        gsl_matrix_set(J, i, 3, Jac[i+(3*m_npts)]);
        gsl_matrix_set(J, i, 4, Jac[i+(4*m_npts)]);
    }
}
/**
 * residuals
 *    Triggers a pointwise parallel residual kernel in the
 *    Device and impedance matches that with gsl's requirements.
 *
 *  @param p  - parameter vector.
 *  @param r  - Residual vector.
 */
void
CudaFitEngine1::residuals(const gsl_vector* p, gsl_vector* r)
{
    float A   = gsl_vector_get(p, P1A_INDEX);
    float k1  = gsl_vector_get(p, P1K1_INDEX);
    float k2  = gsl_vector_get(p, P1K2_INDEX);
    float x1  = gsl_vector_get(p, P1X1_INDEX);
    float C   = gsl_vector_get(p, P1C_INDEX);

    residual1<<<(m_npts+31)/32, 32>>>(
        m_dXtrace, m_dYtrace, m_dResiduals, m_npts,
        C, A, k1, k2, x1
    );
    // Fetch out the residuals and push the minto the r vector:
    
    float resids[m_npts];
    if (hipMemcpy(
        resids, m_dResiduals, m_npts*sizeof(float), hipMemcpyDeviceToHost) !=
        hipSuccess) {
        throwCudaError("Failed to pull residuals from GPU");
    }
    // Push the results into r:
    
    for (int i =0; i < m_npts; i++) {
        gsl_vector_set(r, i, resids[i]);
    }
}
/**
 * throwCudaError
 *    - Find the last cuda error
 *    - Make a string out of the message we're passed and the cuda error.
 *    - throw this all as a runtime_error
 *
 *  @param msg - context message.
 */
void
CudaFitEngine1::throwCudaError(const char* msg)
{
    std::string e="Error: ";
    e += msg;
    e += " : ";
    
    hipError_t status = hipGetLastError();
    e += hipGetErrorString(status);
    
    throw std::runtime_error(e);
}

///////////////////////////////////////////////////////////////////////////
// CudaFitEngine2 implementation - double pulse fits.
//