#include "hip/hip_runtime.h"
/*
    This software is Copyright by the Board of Trustees of Michigan
    State University (c) Copyright 2017.

    You may use this software under the terms of the GNU public license
    (GPL).  The terms of this license are described at:

     http://www.gnu.org/licenses/gpl.txt

     Authors:
             Ron Fox
             Giordano Cerriza
	     NSCL
	     Michigan State University
	     East Lansing, MI 48824-1321
*/

/** 
 * @file  CudaFitEngineAnalytic.cu
 * @brief Provide CUDA fit engines for single- and double-pulse fits.
 * @note  This requires that the CUDA compiler be used.
 * @note  Experimentally the Jacobian for double pulses needs to be double 
 * precision so we've got functions named XXXX which are float and identical 
 * functions named XXXXd which are double.
 */

#include <stdexcept>
#include <math.h>

#include "jacobian_analytic.h"

using namespace ddastoys;
using namespace ddastoys::analyticfit;

// Single pulse fit parameter indices:

static const int P1A_INDEX(0);
static const int P1K1_INDEX(1);
static const int P1K2_INDEX(2);
static const int P1X1_INDEX(3);
static const int P1C_INDEX(4);

// Double pulse fit with all parameters free:

static const int P2A1_INDEX(0);
static const int P2K1_INDEX(1);
static const int P2K2_INDEX(2);
static const int P2X1_INDEX(3);
 
static const int P2A2_INDEX(4);
static const int P2K3_INDEX(5);
static const int P2K4_INDEX(6);
static const int P2X2_INDEX(7);

static const int P2C_INDEX(8);

// From functions_analytic.cpp -> device:

/**
 *
 * @brief Evaluate a logistic function for the specified parameters and point.
 *
 * @details
 * A logistic function is a function with a sigmoidal shape.  We use it
 * to fit the rising edge of signals DDAS digitizes from detectors.
 * See e.g. https://en.wikipedia.org/wiki/Logistic_function for
 * a discussion of this function.
 *
 * @param A  Amplitude of the signal.
 * @param k1 Steepness of the signal (related to the rise time).
 * @param x1 Mid point of the rise of the sigmoid.
 * @param x  Location at which to evaluate the function.
 *
 * @return Logistic function evaluated at x.
 */
__device__
static float
logistic(float A, float k, float x1, float x)
{
    return A/(1+expf(-k*(x-x1)));
}

/**
 * @brief Signals from detectors usually have a falling shape that approximates
 * an exponential. This function evaluates this decay at some point.
 *
 * @param A1 Amplitude of the signal
 * @param k1 Decay time factor f the signal.
 * @param x1 Position of the pulse.
 * @param x  Where to evaluate the signal.
 *
 * @return Value of the exponential decay at x.
 */
__device__
static float
decay(float A, float k, float x1, float x)
{
    return A*(expf(-k*(x-x1)));
}

/**
 * @brief Evaluate the value of a single pulse in accordance with our
 * canonical functional form.  
 * 
 * @details 
 * The form is a sigmoid rise with an exponential decay that sits on top of 
 * a constant offset. The exponential decay is turned on with switchOn() 
 * above when x > the rise point of the sigmoid.
 *
 * @param A1 Pulse amplitiude.
 * @parm  k1 Sigmoid rise steepness.
 * @param k2 Exponential decay time constant.
 * @param x1 Sigmoid position.
 * @param C  Constant offset.
 * @param x  Position at which to evaluat this function
 *
 * @return Single pulse evaluated at x.
 */
__device__
static float
singlePulse(
    float A1, float k1, float k2, float x1, float C, float x
    )
{
    return (logistic(A1, k1, x1, x)  * decay(1.0, k2, x1, x)) + C;
}

/**
 * @brief Evaluate the canonical form of a double pulse.
 *
 * @details 
 * This is done by summing two single pulses. The constant term is thrown 
 * into the first pulse. The second pulse gets a constant term of 0.
 *
 * @param A1 Amplitude of the first pulse.
 * @param k1 Steepness of first pulse rise.
 * @param k2 Decay time of the first pulse.
 * @param x1 Position of the first pulse.
 * @param A2 Amplitude of the second pulse.
 * @param k3 Steepness of second pulse rise.
 * @param k4 Decay time of second pulse.
 * @param x2 Position of second pulse.
 * @param C  Constant offset the pulses sit on.
 * @param x  Position at which to evaluate the pulse.
 *
 * @return Double pulse evaluated at x.
 */
__device__
static float
doublePulse(
    float A1, float k1, float k2, float x1,
    float A2, float k3, float k4, float x2,
    float C, float x    
    )
{
    float p1 = singlePulse(A1, k1, k2, x1, C, x);
    float p2 = singlePulse(A2, k3, k4, x2, 0.0, x);
    
    return p1 + p2;
}

///
// Support functions that are in the device.
//

/**
 * @brief Returns the partial derivative of a single pulse with respect to the
 * amplitude evaluated at a point
 *
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point.
 *
 * @return Value of (dP1/dA)(x)/w
 */
__device__
static float
dp1dA(float k1, float k2, float x1, float x, float w,
      float erise, float efall)
{
    float d = efall;                      // decay(1.0, k2, x1, x);
    float l = 1.0/(1.0 + erise);          // logistic(1.0, k1, x1, x);
    
    return d*l/w;
}
/**
 * @brief Returns the partial derivative of a single pulse with respect to the
 * amplitude evaluated at a point
 *
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point.
 *
 * @return Value of (dP1/dA)(x)/w
 */
__device__
static double
dp1dAd(double k1, double k2, double x1, double x, double w,
       double erise, double efall)
{
    double d = efall;                      // decay(1.0, k2, x1, x);
    double l = 1.0/(1.0 + erise);          // logistic(1.0, k1, x1, x);
    
    return d*l/w;
}

/**
 * @brief Partial of single pulse with respect to the rise time constant k1.
 *
 * @param A  Current guess at amplitude.
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point.
 *
 * @return Value of (dP1/dk1)(x)/w
 */
__device__
static float
dp1dk1(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float d1 =   A*efall;               // decay(A, k2, x1, x);  
    float d2 =   erise; //              // decay(1.0, k1, x1,  x);
    float num = d1*d2*(x - x1);
    float l   =  1.0/(1.0 + erise);     //  logistic(1.0, k1, x1, x);   
    
    return (num*l*l)/w;
}
/**
 * @brief Partial of single pulse with respect to the rise time constant k1.
 *
 * @param A  Current guess at amplitude.
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point.
 *
 * @return Value of (dP1/dk1)(x)/w
 */
__device__
static double
dp1dk1d(double A, double k1, double k2, double x1, double x, double w,
	double erise, double efall)
{
    double d1 =   A*efall;               // decay(A, k2, x1, x);  
    double d2 =   erise; //              // decay(1.0, k1, x1,  x);
    double num = d1*d2*(x - x1);
    double l   =  1.0/(1.0 + erise);     //  logistic(1.0, k1, x1, x);   
    
    return (num*l*l)/w;
}

/**
 * @brief Partial of a single pulse with respect to the decay time constant.

 * @param A  Current guess at amplitude.
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point .
 *
 * @return   Value of (dP1/dk2)(x)/w.
 */
__device__
static float
dp1dk2(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float d1 = A*efall;                   // decay(A, k2, x1, x);
    float num = d1*(x1 - x);
    float l = 1.0/(1.0 + erise);          // logistic(1.0, k1, x1, x);
    
    return (num*l)/w;
}
/**
 * @brief Partial of a single pulse with respect to the decay time constant.

 * @param A  Current guess at amplitude.
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point .
 *
 * @return   Value of (dP1/dk2)(x)/w.
 */
__device__
static double
dp1dk2d(double A, double k1, double k2, double x1, double x, double w,
	double erise, double efall)
{
    double d1 = A*efall;                   // decay(A, k2, x1, x);
    double num = d1*(x1 - x);
    double l = 1.0/(1.0 + erise);          // logistic(1.0, k1, x1, x);
    
    return (num*l)/w;
}

/**
 * @brief Partial of a single pulse with respect to the time at the middle
 * of the pulse's rise.
 *
 * @param A  Current guess at amplitude.
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point.
 * 
 * @return Value of (dP1/dk2)(x)/w.
 */
__device__
static float
dp1dx1(float A, float k1, float k2, float x1, float x, float w,
       float erise, float efall)
{
    float dk1 = erise;                   // decay(1.0, k1, x1, x);
    float dk2 = efall;                   // decay(1.0, k2, x1, x);
    float l   = 1.0/(1.0 + erise);       // logistic(1.0, k1, x1, x);
    
    float left = A*k2*dk2*l;
    float right = A*k1*dk1*dk2*l*l;
    
    return (left - right)/w;
}
/**
 * @brief Partial of a single pulse with respect to the time at the middle
 * of the pulse's rise.
 *
 * @param A  Current guess at amplitude.
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point.
 * 
 * @return Value of (dP1/dk2)(x)/w.
 */
__device__
static double
dp1dx1d(double A, double k1, double k2, double x1, double x, double w,
	double erise, double efall)
{
    double dk1 = erise;                   // decay(1.0, k1, x1, x);
    double dk2 = efall;                   // decay(1.0, k2, x1, x);
    double l   = 1.0/(1.0 + erise);       // logistic(1.0, k1, x1, x);
    
    double left = A*k2*dk2*l;
    double right = A*k1*dk1*dk2*l*l;
    
    return (left - right)/w;
}

/**
 * @brief Partial derivative of single pulse with respect to the constant term
 * evaluated at a point.
 *
 * @param A  Current guess at amplitude.
 * @param k1 Current guess at rise steepness param (log(81)/risetime90).
 * @param k2 Current guess at the decay time constant.
 * @param x1 Current guess at pulse position.
 * @param x  x at which to evaluate all this.
 * @param w  Weight for the point.
 *
 * @return Value of (dP1/dC)(x)/w
 */
__device__
static float
dp1dC(float A, float k1, float k2, float x1, float x, float w)
{
    return 1.0/w;
}


///
// The residual and Jacobian computations are pointwise parallel in the GPU
//

/**
 * @brief Compute the residual for a point in the trace with a single pulse fit.
 *
 * @param tx  Pointer to trace x values.
 * @param ty  Pointer to trace y values.
 * @param res Pointer to residual values.
 * @param len Number of trace elements.
 * @param C   Constant baseline.
 * @param A   Scale factor.
 * @param k1  Rise steepeness.
 * @param k2  Decay time.
 * @param x1  Position.
 */
__global__
void residual1(
    void* tx, void* ty, void* res, unsigned len,
    float C, float A, float k1, float k2, float x1)
{
    // Figure out our index... we just don't do anything if it's
    // bigger than len:    
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
	unsigned short* tracex = static_cast<unsigned short*>(tx);
	unsigned short* tracey = static_cast<unsigned short*>(ty);
	float* resid  = static_cast<float*>(res);
        float x = tracex[i];
        float y = tracey[i];
        
        // Compute the function value.
	// ___device__ function.
        float value = singlePulse(A, k1, k2, x1, C, x);  
        
        // Compute and store the difference:        
        resid[i] = (value - y);        
    }  
}

/**
 * @brief Compute the Jacobian at a single point of the trace for a 
 * single pulse fit.
 *
 * @param tx  Pointer to the trace x coords.
 * @param ty  Pointer to the trace y coords.
 * @param jac Pointer to the Jacobian matrix (len*5 elements)
 * @param len Trace length.
 * @param A   Scale factor.
 * @param k1  Risetime steepeness.
 * @param k2  Decay time.
 * @param x1  Position.
 */
__global__
void jacobian1(
    void* tx, void* ty, void* jac, unsigned len,
    float A, float k1, float k2, float x1
    )
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < len) {
	unsigned short* tracex = static_cast<unsigned short*>(tx);

	float* j = static_cast<float*>(jac);
        float x = tracex[i];
        
        // Common sub-expression elimination:        
        float erise = expf(-k1*(x - x1));
        float efall = expf(-k2*(x - x1));
        
        float dA = dp1dA(k1, k2, x1, x, 1.0, erise, efall);
        float dk1= dp1dk1(A, k1, k2, x1, x, 1.0, erise, efall);
        float dk2= dp1dk2(A, k1, k2, x1, x, 1.0, erise, efall);
        float dx = dp1dx1(A, k1, k2, x1, x, 1.0, erise, efall);
        float dC = dp1dC(A, k1, k2, x1, x, 1.0);
        
        // Put these results in the appropriate Jacobian element:        
        int n = i;
        j[n] = dA;   n += len;
        j[n] = dk1;  n += len;
        j[n] = dk2;  n += len;
        j[n] = dx;   n += len;
        j[n] = dC;
    }
}

///
//  The class implementationon is in the host (CPU).
//

/**
 * @details
 * Allocate the device vectors/matrices. Push the trace x/y points into the 
 * GPU where they stay until we're destroyed.
 */
ddastoys::CudaFitEngine1::CudaFitEngine1(
    std::vector<std::pair<uint16_t, uint16_t>>& data
    ) :
    FitEngine(data)
{
    // Mashall the trace into x/y arrays... this lets them be CUDA memcpied
    // to the GPU    
    m_npts = data.size();    
    unsigned short x[m_npts];
    unsigned short y[m_npts];
    for (int i = 0; i < m_npts; i++) {
        x[i] = data[i].first;
        y[i] = data[i].second;
    }
    
    // The trace:    
    if (hipMalloc(&m_dXtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Failed to allocated X trace points");
    }
    if (hipMalloc(&m_dYtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Failed to allocatee Y trace points");
    }
    if (hipMemcpy(
            m_dXtrace, x, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
	    ) != hipSuccess) {
        throwCudaError("Failed to move trace x coords -> gpu");
    }
    if (hipMemcpy(
            m_dYtrace, y, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
	    ) != hipSuccess) {
        throwCudaError("Failed to move trace y coords -> gpu");
    }
    
    // The residual and jacobians need to be allocated but are filled in by
    // the GPU kernels:    
    if(hipMalloc(&m_dResiduals, m_npts*sizeof(float)) != hipSuccess) {
        throwCudaError("Failed to allocate residual vector");
    }
    if (hipMalloc(&m_dJacobian, m_npts*5*sizeof(float)) != hipSuccess) {
        throwCudaError("Failed to allocated Jacobian");
    }
}

/**
 * @details
 * Just deallocate the GPU resources.
 */
ddastoys::CudaFitEngine1::~CudaFitEngine1()
{
    // Not much point in error checking as we're not going to be able to
    // do anything about errors here anyway.    
    hipFree(m_dXtrace);
    hipFree(m_dYtrace);
    hipFree(m_dResiduals);
    hipFree(m_dJacobian);
}

/**
 * @details
 * We use a Y size of 32 and x size of npts+31/32. That is one warp wide.
 */
void
ddastoys::CudaFitEngine1::jacobian(const gsl_vector* p, gsl_matrix* J)
{
    float A   = gsl_vector_get(p, P1A_INDEX);
    float k1  = gsl_vector_get(p, P1K1_INDEX);
    float k2  = gsl_vector_get(p, P1K2_INDEX);
    float x1  = gsl_vector_get(p, P1X1_INDEX);
    float C   = gsl_vector_get(p, P1C_INDEX);
    
    jacobian1<<<(m_npts+31)/32, 32>>>(
        m_dXtrace, m_dYtrace, m_dJacobian, m_npts, A, k1, k2, x1
	);
    
    if(hipDeviceSynchronize() != hipSuccess) {
	throwCudaError("Synchronizing kernel"); // Block until kernel done.
    }
    
    // Now we need to pull the Jacobian out of the device:    
    float Jac[m_npts*5]; // We'll do it flat
    if(
	hipMemcpy(
	    Jac, m_dJacobian, m_npts*5*sizeof(float), hipMemcpyDeviceToHost
	    ) != hipSuccess
	) {
        throwCudaError("failed to copy Jacobian from device");
    }
    
    // Finally, we have to put the jacobian into the GSL J matrix.    
    for (int i = 0; i < m_npts; i++) {
        gsl_matrix_set(J, i, 0, Jac[i]);
        gsl_matrix_set(J, i, 1, Jac[i+m_npts]);
        gsl_matrix_set(J, i, 2, Jac[i+(2*m_npts)]);
        gsl_matrix_set(J, i, 3, Jac[i+(3*m_npts)]);
        gsl_matrix_set(J, i, 4, Jac[i+(4*m_npts)]);
    }
}

void
ddastoys::CudaFitEngine1::residuals(const gsl_vector* p, gsl_vector* r)
{
    float A   = gsl_vector_get(p, P1A_INDEX);
    float k1  = gsl_vector_get(p, P1K1_INDEX);
    float k2  = gsl_vector_get(p, P1K2_INDEX);
    float x1  = gsl_vector_get(p, P1X1_INDEX);
    float C   = gsl_vector_get(p, P1C_INDEX);

    residual1<<<(m_npts+31)/32, 32>>>(
        m_dXtrace, m_dYtrace, m_dResiduals, m_npts, C, A, k1, k2, x1
	);
    
    if(hipDeviceSynchronize() != hipSuccess) {
	throwCudaError("Synchronizing kernel");	// Block for kernel completion.
    }
    
    // Fetch out the residuals and push the into the r vector:    
    float resids[m_npts];
    if (
	hipMemcpy(
	    resids, m_dResiduals, m_npts*sizeof(float), hipMemcpyDeviceToHost
	    ) != hipSuccess
	) {
        throwCudaError("Failed to pull residuals from GPU");
    }
    
    // Push the results into r:    
    for (int i =0; i < m_npts; i++) {
        gsl_vector_set(r, i, resids[i]);
    }
}

/** 
 * @breif Throw a CUDA error as std::runtime_error.
 *
 * @details
 * - Find the last CUDA error.
 * - Make a string out of the message we're passed and the CUDA error.
 * - Throw this all as a runtime_error.
 *
 *  @param msg Context message.
 */
void
ddastoys::CudaFitEngine1::throwCudaError(const char* msg)
{
    std::string e="Error: ";
    e += msg;
    e += " : ";
    
    hipError_t status = hipGetLastError();
    e += hipGetErrorString(status);
    
    throw std::runtime_error(e);
}

///////////////////////////////////////////////////////////////////////////
// CudaFitEngine2 implementation - double pulse fits.
//

// Device (GPU) kernels needed:

/**
 * @brief Computes the double-pulse residual pointwise parallel.
 *
 * @param xtc  x-coordinates of trace.
 * @param ytc  y-coordinates of trace.
 * @param res  Residuals to compute.
 * @param npts Number of trace points.
 * @param C    Constant offset fit parameter.
 * @param A1   Scale factor for pulse1.
 * @param k11  k1 for pulse 1.
 * @param k12  k2 for pulse 1.
 * @param x1   Position of pulse 1
 * @param A2   Scale factof for pulse 2.
 * @param k21  k1 for pulse 2.
 * @param k22  k2 for pulse 2.
 * @param x2   Position of pulse 2.
 */
__global__
void residual2(
    void* xtc, void* ytc, void* res, unsigned npts,
    float C,
    float A1, float k11, float k12, float x1,
    float A2, float k21, float k22, float x2
    )
{
    // Compute our index and only do anything if its < npts:    
    int i  = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npts) {
	unsigned short* xc = static_cast<unsigned short*>(xtc);
	unsigned short* yc = static_cast<unsigned short*>(ytc);
	float* r = static_cast<float*>(res);
	float x = xc[i];
        float y = yc[i];
        float fit = doublePulse(A1, k11, k12, x1, A2, k21, k22, x2, C, x);
        r[i] = fit - y;
    }
}

/**
 * @brief Compute the double-pulse Jacobian on a point of the pulse. 
 * The Jacobian matrix is an npts x 9 matrix.
 *
 * @param xtc  x-coordinates of the trace.
 * @param jac  Jacobian matrix.
 * @param npts Number of points in the fit.
 * @param A1, k1, k2, x1 Fit parameters for first pulse.
 * @param A2, k3, k4, x2 Fit parameters for the second pulse.
 * @param C    Constant term of the fit.
 */
__global__
void jacobian2(
    void* xtc,  void* jac, unsigned npts,
    double A1, double k1, double k2, double x1,
    double A2, double k3, double k4, double x2,
    double C
    )
{
    // Figure out which point we're doing and compute if it's in the range
    // of the trace:    
    int i  = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < npts) {
	unsigned short* xc = static_cast<unsigned short*>(xtc);
	double* j = static_cast<double*>(jac);
      
        // Now the Jacobian elements:        
        int k = i; // We'll increment this by npts for each j element
        double x = xc[i];
        
        // Common subexpression elmiination between functions:        
        double erise1 = exp(-k1*(x - x1));
        double efall1 = exp(-k2*(x - x1));
        
        double erise2 = exp(-k3*(x - x2));
        double efall2 = exp(-k4*(x - x2));
        
        // Pulse 1 elements:       
        j[k] = dp1dAd(k1, k2, x1, x, 1.0, erise1, efall1);      k += npts;
        j[k] = dp1dk1d(A1, k1, k2, x1, x, 1.0, erise1, efall1); k += npts;
        j[k] = dp1dk2d(A1, k1, k2, x1, x, 1.0, erise1, efall1); k += npts;
        j[k] = dp1dx1d(A1, k1, k2, x1, x, 1.0, erise1, efall1); k += npts;
        
        // Pulse 2 elements:     
        j[k] = dp1dAd(k3, k4,x2,x, 1.0, erise2, efall2);        k += npts;
        j[k] = dp1dk1d(A2, k3, k4, x2, x, 1.0, erise2, efall2); k += npts;
        j[k] = dp1dk2d(A2, k3, k4, x2, x, 1.0, erise2, efall2); k += npts;
        j[k] = dp1dx1d(A2, k3, k4, x2, x, 1.0, erise2, efall2); k += npts;
        
        // Constant element:       
        j[k] = 1.0;
    }
}

////////////////////
// Host class implementation:
//

/**
 * @details
 * Allocate the GPU resources:
 * * Trace x array
 * * Trace y array.
 * * Residual array.
 * * Jacobian vector (m_npts * 9)
 * * Move the trace into the GPU where it stays for all iterations of the fit.
 */
ddastoys::CudaFitEngine2::CudaFitEngine2(
    std::vector<std::pair<uint16_t, uint16_t>>&  data
    ) :
    FitEngine(data)
{
    // Make separate x/y arrays from the data:    
    m_npts = data.size();
    unsigned short x[m_npts];
    unsigned short y[m_npts];
    for (int i =0; i < m_npts; i++) {
	x[i] = data[i].first;
	y[i] = data[i].second;
    }
    
    // Allocate the trace arrays and move the trace in:    
    if (hipMalloc(&m_dXtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Unable to allocate GPU x trace array");
    }
    if (hipMalloc(&m_dYtrace, m_npts*sizeof(unsigned short)) != hipSuccess) {
        throwCudaError("Unable to allocate CPU y trace array");
    }
    
    if(
        hipMemcpy(
            m_dXtrace, x, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
	    ) != hipSuccess
	) {
        throwCudaError("Unable to move x coords of trace -> GPU");
    }
    if(hipMemcpy(
	   m_dYtrace, y, m_npts*sizeof(unsigned short), hipMemcpyHostToDevice
	   ) != hipSuccess ) {
        throwCudaError("Unable to move y coords of trace -> GPU");
    }
    
    // Allocate the residuals and Jacobian:     
    if(hipMalloc(&m_dResiduals, m_npts*sizeof(float)) != hipSuccess) {
        throwCudaError("Unable to allocate residual array in GPU");
    }
    if (hipMalloc(&m_dJacobian, m_npts*9*sizeof(double)) != hipSuccess) {
        throwCudaError("Unable to allocated jacobian matrix in GPU");
    }
}

/**
 * @details
 * Just frees the device blocks.
 */
ddastoys::CudaFitEngine2::~CudaFitEngine2()
{
    // No point in looking for errors since we don't know how to recover:    
    hipFree(m_dXtrace);
    hipFree(m_dYtrace);
    hipFree(m_dResiduals);
    hipFree(m_dJacobian);
}

/**
 * @note We organize the computing into 32 thread blocks because there are 
 * 32 thread per warp.
 */
void
ddastoys::CudaFitEngine2::jacobian(const gsl_vector* p, gsl_matrix* J)
{
    double A1    = gsl_vector_get(p, P2A1_INDEX);   // Pulse 1.
    double k1    = gsl_vector_get(p, P2K1_INDEX);
    double k2    = gsl_vector_get(p, P2K2_INDEX);
    double x1    = gsl_vector_get(p, P2X1_INDEX);
    
    double A2    = gsl_vector_get(p, P2A2_INDEX);   // Pulse 2.
    double k3    = gsl_vector_get(p, P2K3_INDEX);
    double k4    = gsl_vector_get(p, P2K4_INDEX);
    double x2    = gsl_vector_get(p, P2X2_INDEX);
    
    double C     = gsl_vector_get(p, P2C_INDEX);    // constant.
    
    jacobian2<<<(m_npts + 31)/32, 32>>>(
        m_dXtrace, m_dJacobian, m_npts,
        A1, k1, k2, x1,
        A2, k3, k4, x2,
        C
	);
    
    if(hipDeviceSynchronize() != hipSuccess)
	throwCudaError("Failed kernel synchronization");
    
    // Fetch the jacobian and marshall it into j:    
    double jac[m_npts*9];
    if (
	hipMemcpy(
	    jac, m_dJacobian, m_npts*9*sizeof(double), hipMemcpyDeviceToHost
	    ) != hipSuccess
	) {
        throwCudaError("Failed to fetch double-pulse Jacobian from GPU");
    }
    
    for (int i =0; i < m_npts; i++) {
        int k = i;
        gsl_matrix_set(j, i, 0, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 1, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 2, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 3, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 4, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 5, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 6, jac[k]);  k += m_npts;
        gsl_matrix_set(j, i, 7, jac[k]); k += m_npts;
        gsl_matrix_set(j, i, 8, jac[k]); k += m_npts;    
    }
}

void
ddastoys::CudaFitEngine2::residuals(const gsl_vector* p, gsl_vector* r)
{
    // Pull out the current fit parameters:    
    float A1    = gsl_vector_get(p, P2A1_INDEX);   // Pulse 1.
    float k1    = gsl_vector_get(p, P2K1_INDEX);
    float k2    = gsl_vector_get(p, P2K2_INDEX);
    float x1    = gsl_vector_get(p, P2X1_INDEX);
    
    float A2    = gsl_vector_get(p, P2A2_INDEX);   // Pulse 2.
    float k3    = gsl_vector_get(p, P2K3_INDEX);
    float k4    = gsl_vector_get(p, P2K4_INDEX);
    float x2    = gsl_vector_get(p, P2X2_INDEX);
    
    float C     = gsl_vector_get(p, P2C_INDEX);    // constant.
 
    // Fire off the kernel to do all this in pointwise parallel:    
    residual2<<<(m_npts+31)/32,  32>>>(
        m_dXtrace, m_dYtrace, m_dResiduals, m_npts,
        C, 
	A1, k1, k2, x1, 
	A2, k3, k4, x2
	);
    
    if(hipDeviceSynchronize() != hipSuccess)
	throwCudaError("Failed to synchronize kernel");
    
    // Now we pull out the residuals vector and put it into r:    
    float residuals[m_npts];
    if (
	hipMemcpy(
	    residuals, m_dResiduals, m_npts*sizeof(float),
	    hipMemcpyDeviceToHost
	    ) != hipSuccess
	) {
        throwCudaError("Unable to fetch residuals from GPU");
    }
    
    for (int i =0; i < m_npts; i++) {
        gsl_vector_set(r, i, residuals[i]);
    }
}
/**
 * @brief See this method in CudaFitEngine1.
 * 
 * @details
 * Here's a source for factorization into a base class... along with the 
 * allocation of the trace and residual as well as the push of the trace 
 * into the GPU.
 * 
 * @param msg - message used to construct the exception messgae.
 */
void
ddastoys::CudaFitEngine2::throwCudaError(const char* msg)
{
    std::string e="Error: ";
    e += msg;
    e += " : ";
    
    hipError_t status = hipGetLastError();
    e += hipGetErrorString(status);
    
    throw std::runtime_error(e);    
}

